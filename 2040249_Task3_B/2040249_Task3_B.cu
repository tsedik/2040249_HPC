#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

/*
compile with c++ lodepng file:
nvcc -o 2040249_Task3_B 2040249_Task3_B.cu lodepng.cpp 

to run:
./2040249_Task3_B_Task3_B

Tsering Dikey Lama, University of Wolverhampton, Herald College Kathmandu
*/



__device__ unsigned int d_width;

__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4);
	return image[i];
  }
  
  __device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4) +1;
	return image[i];
  }
  
  __device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4) +2;
	return image[i];
  }
  
  __device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4) +3;
	return image[i];
  }
  
  __device__ void setRed(unsigned char *image, unsigned int row, unsigned int col, unsigned char red){
	unsigned int i = (row * d_width * 4) + (col * 4);
	image[i] = red;
  }
  
  __device__ void setGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned char green){
	unsigned int i = (row * d_width * 4) + (col * 4) +1;
	image[i] = green;
  }
  
  __device__ void setBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned char blue){
	unsigned int i = (row * d_width * 4) + (col * 4) +2;
	image[i] = blue;
  }
  
  __device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned char alpha){
	unsigned int i = (row * d_width * 4) + (col * 4) +3;
	image[i] = alpha;
  }
  
__global__ void square(unsigned char * gpu_imageOutput, unsigned char * gpu_imageInput, unsigned int *width){
	
	unsigned redTL, redTC, redTR;
	unsigned redL, redC, redR;
	unsigned redBL, redBC, redBR;
	unsigned newRed;

	unsigned greenTL, greenTC, greenTR;
	unsigned greenL, greenC, greenR;
	unsigned greenBL, greenBC, greenBR;
	unsigned newGreen;

	unsigned blueTL, blueTC, blueTR;
	unsigned blueL, blueC, blueR;
	unsigned blueBL, blueBC, blueBR;
	unsigned newBlue;

	int row = blockIdx.x+1;
	int col = threadIdx.x+1;

	d_width = *width;
	
	setGreen(gpu_imageOutput, row, col, getGreen(gpu_imageInput, row, col));
	setBlue(gpu_imageOutput, row, col, getBlue(gpu_imageInput, row, col));
	setAlpha(gpu_imageOutput, row, col, 255);

	redTL = getRed(gpu_imageInput, row - 1, col - 1);
	redTC = getRed(gpu_imageInput, row - 1, col);
	redTR = getRed(gpu_imageInput, row - 1, col + 1);

	redL = getRed(gpu_imageInput, row, col - 1);
	redC = getRed(gpu_imageInput, row, col);
	redR = getRed(gpu_imageInput, row, col + 1);

	redBL = getRed(gpu_imageInput, row + 1, col - 1);
	redBC = getRed(gpu_imageInput, row + 1, col);
	redBR = getRed(gpu_imageInput, row + 1, col + 1);
	
	//Bluring red color value
	newRed = (redTL+redTC+redTR+redL+redC+redR+redBL+redBC+redBR)/9;  

	setRed(gpu_imageOutput, row, col, newRed);

	greenTL = getGreen(gpu_imageInput, row - 1, col - 1);
	greenTC = getGreen(gpu_imageInput, row - 1, col);
	greenTR = getGreen(gpu_imageInput, row - 1, col + 1);

	greenL = getGreen(gpu_imageInput, row, col - 1);
	greenC = getGreen(gpu_imageInput, row, col);
	greenR = getGreen(gpu_imageInput, row, col + 1);

	greenBL = getGreen(gpu_imageInput, row + 1, col - 1);
	greenBC = getGreen(gpu_imageInput, row + 1, col);
	greenBR = getGreen(gpu_imageInput, row + 1, col + 1);

	//Bluring green color value
	newGreen = (greenTL+greenTC+greenTR+greenL+greenC+greenR+greenBL+greenBC+greenBR)/9; 

	setGreen(gpu_imageOutput, row, col, newGreen);

	blueTL = getBlue(gpu_imageInput, row - 1, col - 1);
	blueTC = getBlue(gpu_imageInput, row - 1, col);
	blueTR = getBlue(gpu_imageInput, row - 1, col + 1);

	blueL = getBlue(gpu_imageInput, row, col - 1);
	blueC = getBlue(gpu_imageInput, row, col);
	blueR = getBlue(gpu_imageInput, row, col + 1);

	blueBL = getBlue(gpu_imageInput, row + 1, col - 1);
	blueBC = getBlue(gpu_imageInput, row + 1, col);
	blueBR = getBlue(gpu_imageInput, row + 1, col + 1);

	//Bluring blue color value
	newBlue = (blueTL+blueTC+blueTR+blueL+blueC+blueR+blueBL+blueBC+blueBR)/9; 

	setBlue(gpu_imageOutput, row, col, newBlue);
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
	  long long int ds =  finish->tv_sec - start->tv_sec; 
	  long long int dn =  finish->tv_nsec - start->tv_nsec; 

	  if(dn < 0 ) 
	  {
	    ds--;
	    dn += 1000000000; 
          } 

	  *difference = ds * 1000000000 + dn;
	  return !(*difference > 0);
}

int main(int argc, char **argv){

	struct timespec start, finish;
	long long int time_elapsed;

	unsigned char *image;
	unsigned int width;
	unsigned int height;
	const char* filename = "hck.png";
	const char* newFileName = "filtered.png";
	
	clock_gettime(CLOCK_MONOTONIC, &start);//Start monitoring the duration 

	//Decoding Image
	lodepng_decode32_file(&image, &width, &height, filename);

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	//Declaring gpuImageWidth and setting the value 
	unsigned int* d_wid; 
	hipMalloc( (void**) &d_wid, sizeof(int));
	hipMemcpy(d_wid, &width, sizeof(int), hipMemcpyHostToDevice);

	// launch the kernel
	square<<<height-1, width-1>>>(d_out, d_in, d_wid);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	//Encoding Image
	lodepng_encode32_file(newFileName, host_imageOutput, width, height);

	hipFree(d_in);
	hipFree(d_out);
	clock_gettime(CLOCK_MONOTONIC, &finish); //End the duration of the program
	
	//Calculate difference
	time_difference(&start, &finish, &time_elapsed);
	

	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, 
         (time_elapsed/1.0e9)); 

	return 0;
}




