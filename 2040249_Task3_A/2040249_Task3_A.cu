
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*
	to compile :
	nvcc -o 2040249_Task3_A 2040249_Task3_A.c 

	to run:
	./2040249_Task3_A

	Tsering Dikey Lama, University of Wolverhampton, Herald College Kathmandu
*/


__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;  
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;  
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword; //Returns encrypted password
}

__device__ int comparePass(char* str1, char* str2){
	
    while(*str1)
    {
        
        if (*str1 != *str2)
            break;
 
        //Changing Pointer location
        str1++;
        str2++;
    }
 
    // Returing the 0 if the two strings matches 
    return *(const unsigned char*)str1 - *(const unsigned char*)str2;
}

__global__ void crack(char * alphabet, char * numbers){

	char genRawPass[4];
	//Adding test passwords to genRawPass
	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];

	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];

	char password[] = "td02";
	//Raw Password being encrypted
	char *encPassword = CudaCrypt(password);
	
	//Comparing encrypted genRawPass with encPassword
	if(comparePass(CudaCrypt(genRawPass),encPassword) == 0){
		printf("Your password is cracked : %s = %s\n", genRawPass, password);
	}
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 

	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}


int main(int argc, char ** argv){

	char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
	char cpuNumbers[10] = {'0','1','2','3','4','5','6','7','8','9'};

	char * gpuAlphabet;
	hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
	hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

	char * gpuNumbers;
	hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10); 
	hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

	char * password;
	hipMalloc( (void**) &password, sizeof(char) * 26); 
	hipMemcpy(password, argv[1], sizeof(char) * 26, hipMemcpyHostToDevice);

	struct timespec start, finish;
	long long int time_elapsed;
	
	//Start monitoring the duration 
	clock_gettime(CLOCK_MONOTONIC, &start);
	
	crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers);
	hipDeviceSynchronize();

	//End the duration of the program
	clock_gettime(CLOCK_MONOTONIC, &finish);
	
	//Calculate the duration
	time_difference(&start, &finish, &time_elapsed);
	
	//Print the duration taken
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
	(time_elapsed/1.0e9)); 
	
	return 0;
}


	











